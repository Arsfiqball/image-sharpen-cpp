#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include "bitmap.h"

using namespace std;

/*
 * NOTE!
 *
 * This only works with 24-bit based uncompressed Bitmap format.
 * Use this tool below to convert your image into compatible format.
 * https://online-converting.com/image/convert2bmp/
 * also, don't forget to choose "Color" option to "24 Bit (True Color)."
 *
 * The algorithm is based on this tutorial (with few changes)
 * https://lodev.org/cgtutor/filtering.html#Sharpen
 *
 */

#define filterWidth 5
#define filterHeight 5
#define factor 0.125
#define bias 0.0

int filter[filterHeight * filterWidth] =
{
  -1, -1, -1, -1, -1,
  -1,  2,  2,  2, -1,
  -1,  2,  8,  2, -1,
  -1,  2,  2,  2, -1,
  -1, -1, -1, -1, -1,
};

__global__ void kRunFilter (int h, int w, int* filter, int* red, int* green, int* blue, int* outRed, int* outGreen, int* outBlue) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  double newRed = 0.0;
  double newGreen = 0.0;
  double newBlue = 0.0;

  if (y < h && x < w) {
    for (int filterY = 0; filterY < filterHeight; filterY++) {
      for (int filterX = 0; filterX < filterWidth; filterX++) {
        int imageX = (x - int(filterWidth / 2) + filterX + w) % w;
        int imageY = (y - int(filterHeight / 2) + filterY + h) % h;
        newRed += red[imageY * w + imageX] * filter[filterY * filterWidth + filterX];
        newGreen += green[imageY * w + imageX] * filter[filterY * filterWidth + filterX];
        newBlue += blue[imageY * w + imageX] * filter[filterY * filterWidth + filterX];
      }
    }

    outRed[y * w + x] = min(max(int(factor * newRed + bias), 0), 255);
    outGreen[y * w + x] =  min(max(int(factor * newGreen + bias), 0), 255);
    outBlue[y * w + x] =  min(max(int(factor * newBlue + bias), 0), 255);
  }
}

int main () {
  Bitmap image;
  vector <vector <Pixel> > bmp;

  image.open("image_source.bmp");

  bool validBmp = image.isImage();

  cout << "start" << endl;

  if (validBmp == true) {
    bmp = image.toPixelMatrix();
    int h = bmp.size();
    int w = bmp[0].size();

    size_t matrixImageSize = h * w * sizeof(int);
    size_t matrixFilterSize = filterWidth * filterHeight * sizeof(int);

    int *h_red = (int*)malloc(matrixImageSize);
    int *h_green = (int*)malloc(matrixImageSize);
    int *h_blue = (int*)malloc(matrixImageSize);

    cout << "creating rgb" << endl;

    for (int i = 0; i < h; i++) {
      for (int j = 0; j < w; j++) {
        *(h_red + i * w + j) = (int) bmp[i][j].red;
        *(h_green + i * w + j) = (int) bmp[i][j].green;
        *(h_blue + i * w + j) = (int) bmp[i][j].blue;
      }
    }

    cout << "copying memories" << endl;

    int *d_red;
    int *d_green;
    int *d_blue;
    int *d_outRed;
    int *d_outGreen;
    int *d_outBlue;
    int *d_filter;

    hipMalloc((void**)&d_red, matrixImageSize);
    hipMalloc((void**)&d_green, matrixImageSize);
    hipMalloc((void**)&d_blue, matrixImageSize);
    hipMalloc((void**)&d_outRed, matrixImageSize);
    hipMalloc((void**)&d_outGreen, matrixImageSize);
    hipMalloc((void**)&d_outBlue, matrixImageSize);
    hipMalloc((void**)&d_filter, matrixFilterSize);

    hipMemcpy(d_red, h_red, matrixImageSize, hipMemcpyHostToDevice);
    hipMemcpy(d_green, h_green, matrixImageSize, hipMemcpyHostToDevice);
    hipMemcpy(d_blue, h_blue, matrixImageSize, hipMemcpyHostToDevice);
    hipMemcpy(d_filter, filter, matrixFilterSize, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((int) ceil(1024 / threadsPerBlock.x), (int) ceil(1024 / threadsPerBlock.y));

    kRunFilter <<< numBlocks, threadsPerBlock >>> (
      h,
      w,
      d_filter,
      d_red,
      d_green,
      d_blue,
      d_outRed,
      d_outGreen,
      d_outBlue
    );

    hipDeviceSynchronize();

    hipMemcpy(h_red, d_outRed, matrixImageSize, hipMemcpyDeviceToHost);
    hipMemcpy(h_green, d_outGreen, matrixImageSize, hipMemcpyDeviceToHost);
    hipMemcpy(h_blue, d_outBlue, matrixImageSize, hipMemcpyDeviceToHost);

    for (int i = 0; i < h; ++i) {
      for (int j = 0; j < w; ++j) {
        bmp[i][j].red = h_red[i * w + j];
        bmp[i][j].green = h_green[i * w + j];
        bmp[i][j].blue = h_blue[i * w + j];
      }
    }

    hipFree(d_filter);
    hipFree(d_red);
    hipFree(d_green);
    hipFree(d_blue);

    free(h_red);
    free(h_green);
    free(h_blue);

    image.fromPixelMatrix(bmp);
    image.save("output.bmp");
  }

  cout << "done" << endl;

  return 0;
}
